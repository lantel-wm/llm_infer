#include "hip/hip_runtime.h"
#include "embedding_kernel_gpu.cuh"

namespace kernel {
__global__ void embedding_kernel_gpu_fp32(int32_t vocab_size, int32_t token_num,
                                          int32_t embedding_dim, const int32_t* input_ptr,
                                          const float* weight_ptr, float* output_ptr) {
  for (int32_t token_idx = blockIdx.x; token_idx < token_num; token_idx += gridDim.x) {
    if (token_idx >= token_num) {
      return;
    }
    int32_t token = input_ptr[token_idx];
    if (token >= vocab_size) {
      return;
    }

    // output: [seq_len, embedding_dim]
    // weight: [vocab_size, embedding_dim]
    float* output_ptr_start = output_ptr + token_idx * embedding_dim;
    const float* weight_ptr_start = weight_ptr + token * embedding_dim;
    // threadIdx.x :  0, 1, 2, .., 127
    // blockDim.x  :  128
    for (int32_t i = threadIdx.x; i < embedding_dim; i += blockDim.x) {
      output_ptr_start[i] = weight_ptr_start[i];
    }
  }
}

void embedding_kernel_gpu(const tensor::Tensor& input, const tensor::Tensor& weight,
                          const tensor::Tensor& output, int32_t vocab_size, void* stream) {
  CHECK(!input.is_empty());
  CHECK(!weight.is_empty());
  CHECK(!output.is_empty());

  CHECK(input.device_type() == core::DeviceType::GPU &&
        weight.device_type() == core::DeviceType::GPU &&
        output.device_type() == core::DeviceType::GPU);

  const int32_t token_num = static_cast<int32_t>(input.size());
  const int32_t embedding_dim = weight.get_dim(1);

  constexpr int32_t grid_size = 512;
  const int32_t block_size = 256;
  int32_t* in_ptr = const_cast<int32_t*>(input.ptr<int32_t>());
  float* wei_ptr = const_cast<float*>(weight.ptr<float>());
  float* out_ptr = const_cast<float*>(output.ptr<float>());
  if (stream) {
    hipStream_t stream_ = static_cast<hipStream_t>(stream);
    embedding_kernel_gpu_fp32<<<grid_size, block_size, 0, stream_>>>(
        vocab_size, token_num, embedding_dim, in_ptr, wei_ptr, out_ptr);
  } else {
    embedding_kernel_gpu_fp32<<<grid_size, block_size>>>(vocab_size, token_num, embedding_dim,
                                                         in_ptr, wei_ptr, out_ptr);
  }
}
}  // namespace kernel