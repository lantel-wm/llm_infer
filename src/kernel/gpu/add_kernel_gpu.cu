#include "hip/hip_runtime.h"
#include "add_kernel_gpu.cuh"

namespace kernel {

__global__ void add_kernel_gpu_fp32(int32_t size, const float* in1, const float* in2, float* out) {
  int32_t tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid >= size) {
    return;
  }
  float in_val1 = in1[tid];
  float in_val2 = in2[tid];
  out[tid] = in_val1 + in_val2;
}

void add_kernel_gpu(const tensor::Tensor& input1, const tensor::Tensor& input2,
                    const tensor::Tensor& output, void* stream) {
  CHECK_EQ(input1.is_empty(), false);
  CHECK_EQ(input2.is_empty(), false);
  CHECK_EQ(output.is_empty(), false);
  int32_t size = static_cast<int32_t>(input1.size());
  CHECK_EQ(size, input2.size());
  CHECK_EQ(size, output.size());
  int32_t block_size = 512;
  int32_t grid_size = (size + block_size - 1) / block_size;
  if (stream) {
    hipStream_t stream_ = static_cast<ihipStream_t*>(stream);
    add_kernel_gpu_fp32<<<grid_size, block_size, 0, stream_>>>(
        size, input1.ptr<float>(), input2.ptr<float>(), const_cast<float*>(output.ptr<float>()));
  } else {
    add_kernel_gpu_fp32<<<grid_size, block_size>>>(size, input1.ptr<float>(), input2.ptr<float>(),
                                                   const_cast<float*>(output.ptr<float>()));
  }
}
}  // namespace kernel
